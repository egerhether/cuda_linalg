#include "cuda_utils.cuh"
#include <string>

void cuda_check(hipError_t err)
{
    if (err != hipSuccess)
        throw std::string(hipGetErrorString(err));
}

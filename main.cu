#include "hip/hip_runtime.h"
#include "linalg/matrix.cuh"
#include <chrono>
#include <iostream>
#include <string>

void test_transpose(int size)
{
    linalg::Matrix mat(size, size, 1.0);
    std::cout << "Tranpose benchmark\n";
    // gpu
    mat.gpu();
    auto begin = std::chrono::steady_clock::now();
    linalg::Matrix transpose = mat.transpose();
    auto end = std::chrono::steady_clock::now();
    auto time_diff = std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count();
    std::cout << "[GPU] Transposing matrix of size " << size << " took " << time_diff << " ms.\n";

    // cpu
    mat.cpu();
    begin = std::chrono::steady_clock::now();
    linalg::Matrix transpose_cpu = mat.transpose();
    end = std::chrono::steady_clock::now();
    time_diff = std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count();
    std::cout << "[CPU] Transposing matrix of size " << size << " took " << time_diff << " ms.\n";

    // difference
    linalg::Matrix diff = transpose - transpose_cpu;
    float err = diff.norm();
    std::cout << "Difference between results: " << err << "\n\n";
}

void test_addition(int size)
{
    linalg::Matrix mat(size, size, 1.0);
    // bechmark addition
    std::cout << "Addition benchmark\n";
    linalg::Matrix to_add(size, size, 4.3);
    // gpu
    to_add.gpu();
    mat.gpu();
    auto begin = std::chrono::steady_clock::now();
    linalg::Matrix sum = mat + to_add;
    auto end = std::chrono::steady_clock::now();
    auto time_diff = std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count();
    std::cout << "[GPU] Matrix addition of size " << size << " took " << time_diff << " ms.\n";

    // cpu
    to_add.cpu();
    mat.cpu();
    begin = std::chrono::steady_clock::now();
    linalg::Matrix sum_cpu = mat + to_add;
    end = std::chrono::steady_clock::now();
    time_diff = std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count();
    std::cout << "[CPU] Matrix addition of size " << size << " took " << time_diff << " ms.\n";

    // difference
    linalg::Matrix diff = sum - sum_cpu;
    float err = diff.norm();
    std::cout << "Difference between results: " << err << "\n\n";
}

void test_matmul(int size)
{
    linalg::Matrix mat(size, size, 1.0);
    // benchmark matmul
    std::cout << "Matmul benchmark\n";
    // smaller matrices as it is a heavier operation
    linalg::Matrix to_mult(size / 5, size / 5, 3.2);
    linalg::Matrix mult_mat = linalg::Matrix(size / 5, size / 5, 1.0);
    // gpu
    to_mult.gpu();
    mult_mat.gpu();
    auto begin = std::chrono::steady_clock::now();
    linalg::Matrix prod = mult_mat * to_mult;
    auto end = std::chrono::steady_clock::now();
    auto time_diff = std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count();
    std::cout << "[GPU] Matrix multiplication of size " << size / 5 << " took " << time_diff << " ms.\n";

    // cpu
    to_mult.cpu();
    mult_mat.cpu();
    begin = std::chrono::steady_clock::now();
    linalg::Matrix prod_cpu = mult_mat * to_mult;
    end = std::chrono::steady_clock::now();
    time_diff = std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count();
    std::cout << "[CPU] Matrix multiplication of size " << size / 5 << " took " << time_diff << " ms.\n";

    // difference
    linalg::Matrix diff = prod - prod_cpu;
    float err = diff.norm();
    std::cout << "Difference between results: " << err << "\n\n";
}

void test_inverse(int size)
{
    linalg::Matrix mat(size, size, 1.0);
    // benchmark inverse
    std::cout << "Inverse benchmark\n";
    linalg::Matrix to_inv(size / 5, size / 5, 1.0);
    to_inv.fill_random();
    // gpu
    to_inv.gpu();
    auto begin = std::chrono::steady_clock::now();
    linalg::Matrix inverse = to_inv.inv();
    auto end = std::chrono::steady_clock::now();
    auto time_diff = std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count();
    std::cout << "[GPU] Matrix inverse of size " << size / 5 << " took " << time_diff << " ms.\n";

    // cpu
    to_inv.cpu();
    begin = std::chrono::steady_clock::now();
    linalg::Matrix inverse_cpu = to_inv.inv();
    end = std::chrono::steady_clock::now();
    time_diff = std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count();
    std::cout << "[CPU] Matrix inverse of size " << size / 5 << " took " << time_diff << " ms.\n";

    // difference
    linalg::Matrix diff = inverse - inverse_cpu;
    inverse.print();
    linalg::Matrix id = inverse * to_inv;
    id.print();
    float err = diff.norm();
    std::cout << "Difference between results: " << err << "\n\n";
}

int main(int argc, char **argv)
{
    try {
        int size = 10000;

        if (argc != 1 && strcmp(argv[1], "-d") == 0) {
            test_inverse(size);
            return 0;
        }

        if (argc != 1)
            size = std::stoi(argv[1]);

        test_transpose(size);

        test_addition(size);

        test_matmul(size);

        test_inverse(size);

    } catch (std::string e) {

        std::cout << e << '\n';
    } catch (std::logic_error e) {
        std::cout << e.what() << '\n';
    }
}
